#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <string.h>
#include "error.h"


// 1. Check how many compute devices are attached.
// 2. Select device with atleast compute capability 1.3.
int main() {
  int id;                     // 1
  TRY( hipGetDevice(&id) );  // 1
  printf("Current CUDA device: %d\n", id);

  hipDeviceProp_t p;                 // 2
  memset(&p, 0, sizeof(p));         // 2  
  p.major = 1;                      // 2
  p.minor = 3;                      // 2
  TRY( hipChooseDevice(&id, &p) ); // 2
  printf("CUDA device with atleast compute capability 1.3: %d\n\n", id);
  printf("Cards that have compute capability 1.3 or higher\n"
         "support double-precision floating-point math.\n");
  TRY( hipSetDevice(id) );         // 2
  return 0;
}
